
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 16

__global__ void set(int *A, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;    	//index saved
    A[idx] = idx;                                       	//A[1] = 1, A[2] = 2, ..., A[N] = N
}

int main(void)
{
    const int N = 128;                                  	//number of steps
    int size =  N * sizeof(int);                        	//to clean up code

    int *d_A;                                           	//device array
    int *h_A;                                           	//host array

    h_A = (int*) malloc(size);                          	//allocate memory to host array
    hipMalloc((void **) &d_A, size);                   	//allocate memory to device array

    set<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_A, N);  	//launch set kernel on GPU

    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost); 	//copy device array to host array

    for(int i = 0; i <= N; i++)
    {
	printf("%d ", h_A[i]);                               	//print host array

	if(i%16 == 0 && i != 0 || i == N)
	{
	    printf("\n");
	}
    }

    free(h_A);                                          	//free host array
    hipFree(d_A);                                      	//free device arrray

    return 0;
}
